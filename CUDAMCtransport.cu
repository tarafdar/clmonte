#include "hip/hip_runtime.h"
/*	This file is part of CUDAMC.

    CUDAMC is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    CUDAMC is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with CUDAMC.  If not, see <http://www.gnu.org/licenses/>.*/

__global__ void MCd(unsigned int* xd,unsigned int* cd, unsigned int* ad,unsigned int* numd,unsigned int* histd)
{

    //for loops
    unsigned int ii=0;

    //First element processed by the block
    //int begin=NUM_THREADS_PER_BLOCK*bx;

    unsigned long long int x=cd[NUM_THREADS_PER_BLOCK*blockIdx.x+threadIdx.x];
	
	x=(x<<32)+xd[NUM_THREADS_PER_BLOCK*blockIdx.x+threadIdx.x];


    unsigned int a=ad[NUM_THREADS_PER_BLOCK*blockIdx.x+threadIdx.x];


	float3 pos; //float triplet to store the position
	float3 dir; //float triplet to store the direction
	float t;	//float to store the time of flight
	float s;	//step length
	
	float mus_max=90.0f;	//[1/cm]
	float v=0.0214f;		//[cm/ps] (c=0.03 [cm/ps] v=c/n) here n=1.4
	float cos_crit=0.6999f;	//the critical angle for total internal reflection at the border cos_crit=sqrt(1-(nt/ni)^2)
	float g=0.9f;	
	float n=1.4f;

	unsigned int num_det_photons=0;
	unsigned int flag=0;

	LaunchPhoton(&pos, &dir, &t);//Launch the photon
	
	for(ii=0;ii<NUMSTEPS_GPU;ii++) //this is the main while loop
	{
		//num_det_photons++;
		s = __fdividef(-__logf(rand_MWC_oc(&x,&a)),mus_max);//sample step length 
		
		//Perform boundary crossing check here
		if((pos.z+dir.z*s)<=0)//photon crosses boundary within the next step
		{
			flag=Reflect(&dir,&pos,&t,&v,&cos_crit,&n,&x,&a,histd);
		}
		
		//Move (we can move the photons that have been terminated above since it improves our performance and does not affect our results)
		pos.x += s*dir.x;
		pos.y += s*dir.y;
		pos.z += s*dir.z;
		t += __fdividef(s,v); 

		Spin(&dir,&g,&x,&a);

		if(t >= TMAX || flag>=1)//Kill photon and launch a new one
		{
			num_det_photons++;
			flag=0;
			LaunchPhoton(&pos, &dir, &t);//Launch the photon
		}
		

	}//end main for loop!
	

	__syncthreads();//necessary?

	numd[NUM_THREADS_PER_BLOCK*blockIdx.x+threadIdx.x]/*[begin+tx]*/=num_det_photons; 

}//end MCd

__device__ float rand_MWC_co(unsigned long long* x,//unsigned int* c,
		       unsigned int* a)
{
		//Generate a random number [0,1)
		//this implementation seems to be faster
		*x=(*x&0xffffffffull)*(*a)+(*x>>32);
		return((float)((unsigned int)(*x&0xffffffffull))/(UINT_MAX));

}//end __device__ rand_MWC_co

__device__ float rand_MWC_oc(unsigned long long* x,//unsigned int* c,
		       unsigned int* a)
{
		//Generate a random number (0,1]
		*x=(*x&0xffffffffull)*(*a)+(*x>>32);
		return(1.0f-(float)((unsigned int)(*x&0xffffffffull))/(UINT_MAX));
}//end __device__ rand_MWC_oc


__device__ void LaunchPhoton(float3* pos, float3* dir, float* t)
{
	pos->x=0.0f;
	pos->y=0.0f;
	pos->z=0.0f;

	dir->x=0.0f;
	dir->y=0.0f;
	dir->z=1.0f;

	*t=0.0f;
}



__device__ void Spin(float3* dir, float* g, unsigned long long* x,//unsigned int* c,
		       unsigned int* a)
{
	float cost, sint;	// cosine and sine of the 
						// polar deflection angle theta. 
	float cosp, sinp;	// cosine and sine of the 
						// azimuthal angle psi. 
	float temp;

	float tempdir=dir->x;


	//This is more efficient for g!=0 but of course less efficient for g==0
	temp = __fdividef((1.0f-(*g)*(*g)),(1.0f-(*g)+2.0f*(*g)*rand_MWC_co(x,a)));//Should be close close????!!!!!
	cost = __fdividef((1.0f+(*g)*(*g) - temp*temp),(2.0f*(*g)));
	if((*g)==0.0f)
		cost = 2.0f*rand_MWC_co(x,a) -1.0f;


	sint = sqrtf(1.0f - cost*cost);

	__sincosf(2.0f*PI*rand_MWC_co(x,a),&cosp,&sinp);
	

	temp = sqrtf(1.0f - dir->z*dir->z);

	if(temp==0.0f)// normal incident.
	{
		dir->x = sint*cosp;
		dir->y = sint*sinp;
		dir->z = copysignf(cost,dir->z*cost);
	}
	else // regular incident.
	{
		dir->x = __fdividef(sint*(dir->x*dir->z*cosp - dir->y*sinp),temp) + dir->x*cost;
		dir->y = __fdividef(sint*(dir->y*dir->z*cosp + tempdir*sinp),temp) + dir->y*cost;
		dir->z = -sint*cosp*temp + dir->z*cost;
	}

	//normalisation seems to be required as we are using floats! Otherwise the small numerical error will accumulate
	temp=rsqrtf(dir->x*dir->x+dir->y*dir->y+dir->z*dir->z);
	dir->x=dir->x*temp;
	dir->y=dir->y*temp;
	dir->z=dir->z*temp;
	
}


__device__ unsigned int Reflect(float3* dir, float3* pos, float* t, float* v, float* cos_crit, float* n, unsigned long long* x,//unsigned int* c,
		       unsigned int* a,unsigned int* histd)
{
	float r;
	float fibre_separtion=1.0f;//[cm]
	float fibre_diameter=0.05f;//[cm]

	if(-dir->z<=*cos_crit)
		r=1.0f; //total internal reflection
	else
	{
		if(-dir->z==1.0f)//normal incident
		{		
			r = __fdividef((1.0f-*n),(1+*n));
			r *= r;//square
		}
		else
		{
			//long and boring calculations of r
			float sinangle_i = sqrtf(1.0f-dir->z*dir->z);
			float sinangle_t = *n*sinangle_i;
			float cosangle_t = sqrtf(1.0f-sinangle_t*sinangle_t);
			
			float cossumangle = (-dir->z*cosangle_t) - sinangle_i*sinangle_t;
			float cosdiffangle = (-dir->z*cosangle_t) + sinangle_i*sinangle_t;
			float sinsumangle = sinangle_i*cosangle_t + (-dir->z*sinangle_t);
			float sindiffangle = sinangle_i*cosangle_t - (-dir->z*sinangle_t); 
			
			r = 0.5*sindiffangle*sindiffangle*__fdividef((cosdiffangle*cosdiffangle+cossumangle*cossumangle),(sinsumangle*sinsumangle*cosdiffangle*cosdiffangle));
		
		}
	}
	if(r<1.0f)
	{
		if(rand_MWC_co(x/*,c*/,a)<=r)//reflect
			r=1.0f;
		else//transmitt
		{
			//calculate x and y where the photon escapes the medium
			
			r=__fdividef(pos->z,-dir->z);//dir->z must be finite since we have a boundary cross!
			pos->x+=dir->x*r;
			pos->y+=dir->y*r;
			*t+=__fdividef(r,*v); //calculate the time when the photon exits

			r=sqrtf(pos->x*pos->x+pos->y*pos->y);
			
			//check for detection here
			if(fabsf(r-fibre_separtion)<=fibre_diameter)
			{
				//photon detected!
-				atomicAdd( histd + __float2uint_rz(__fdividef((*t),DT)) , 1);//&histd[(unsigned int)floorf(__fdividef((t*),DT))],(unsigned int)1);
				return 1;
			}
			else
			{
				return 2;
			}	
		}
	}
	if(r==1.0f)//reflect (mirror z and dz in reflection plane)
	{
		pos->z *= -1;//mirror the z-coordinate in the z=0 plane, equal to a reflection.
		dir->z *= -1;// do the same to the z direction vector
	}
	return 0;
}
