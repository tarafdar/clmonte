#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////
//
//		CUDA-based Monte Carlo simulation of photon migration in semi infinite media.
//	
//			This is the version of the code used in the letter submitted to JBO-letters 2008
//			Currently the code is in an experimental state, i.e. the code is not always pretty 
//			or efficient and some ways of implementing certain aspects of the code are far 
//			from desirable. Still it should provide a good starting point for anyone interested 
//			in CUDA-based Monte Carlo simulations of photon migration. 
//
//			For the JBO-letters article the code was run on a NVIDIA 8800GT and the number of 
//			threads are hence optimized for this particular card.
//
//			We apologize for the lack of comment in the current code. We will soon re-relese 
//			this code with detailed explanations of the implementation as well as proper commenting.
//
//			To compile and run this code, please visit www.nvidia.com and download the necessary 
//			CUDA Toolkit and SKD. I also highly recommend the Visual Studio wizard 
//			(available at:http://forums.nvidia.com/index.php?showtopic=69183) 
//			if you use Visual Studio 2005 
//			(The express edition is available for free at: http://www.microsoft.com/express/2005/). 
//
//			This code is distributed under the terms of the GNU General Public Licence (see
//			below). If you use this code for academic purposes, we would greatly appreciate a 
//			citation of our letter describing GPU-based Monte Carlo simulations of photon migration. 
//
//
///////////////////////////////////////////////////////////////

/*	This file is part of CUDAMC.

    CUDAMC is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    CUDAMC is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with CUDAMC.  If not, see <http://www.gnu.org/licenses/>.*/


//#define NUM_THREADS_PER_BLOCK 320 //Keep above 192 to eliminate global memory access overhead
//#define NUM_THREADS_PER_BLOCK 896 //Keep above 192 to eliminate global memory access overhead
#define NUM_THREADS_PER_BLOCK 560 //Keep above 192 to eliminate global memory access overhead
#define NUM_BLOCKS 48 //Keep numblocks a multiple of the #MP's of the GPU (8800GT=14MP)
//#define NUM_BLOCKS 30 //Keep numblocks a multiple of the #MP's of the GPU (8800GT=14MP)
#define NUM_THREADS 26880
#define NUMSTEPS_GPU 500000
#define NUMSTEPS_CPU 500000
#define PI 3.14159265f

#define TMAX 2000.0f //[ps] Maximum time of flight
#define DT 10.0f //[ps] Time binning resolution
#define TEMP 201 //ceil(TMAX/DT), precalculated to avoid dynamic memory allocation (fulhack)


unsigned int xtest[NUM_THREADS];
unsigned int ctest[NUM_THREADS];
unsigned int atest[NUM_THREADS];

// forward declaration of the device code
__global__ void MCd(unsigned int*,unsigned int*,unsigned int*,unsigned int*,unsigned int*);
__device__ float rand_MWC_oc(unsigned long long*,unsigned int*);
__device__ float rand_MWC_co(unsigned long long*,unsigned int*);
__device__ void LaunchPhoton(float3*, float3*, float*);
__device__ void Spin(float3*,float*,unsigned long long*,unsigned int*);
__device__ unsigned int Reflect(float3*, float3*, float*, float*, float*, float*, unsigned long long*,unsigned int*,unsigned int*);

// forward declaration of the host code
void MCh(unsigned int*,unsigned int*,unsigned int*,unsigned int*,unsigned int*);
float rand_MWC_och(unsigned long long*,unsigned int*);
float rand_MWC_coh(unsigned long long*,unsigned int*);
void LaunchPhotonh(float3*, float3*, float*);
void Spinh(float3*,float*,unsigned long long*,unsigned int*);
unsigned int Reflecth(float3*, float3*, float*, float*, float*, float*, unsigned long long*,unsigned int*,unsigned int*);
/*
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include "hip/device_functions.h"
// helper functions and utilities to work with CUDA
//#include "helper_functions.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include "math_functions.h"
#include "common_functions.h"
//#include "sm_11_atomic_functions.h"
//#include "sm_35_atomic_functions.h"
*/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
//include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
//include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

//#include "sm_12_atomic_functions.h"
//include "sm_13_double_functions.h"
#include "hip/hip_math_constants.h"
#include <stdio.h>
//#include "cutil_math.h"
#include "CUDAMCtransport.cu"
#include "CUDAMC_goldstandard.c"
#include <time.h>


// wrapper for device code
void MC(unsigned int* x,unsigned int* c,unsigned int* a)
{
	unsigned int num[NUM_THREADS];
	unsigned long long num_tot, hist_tot;
	unsigned int i;
	unsigned int hist[TEMP];

	unsigned int numh[NUM_THREADS];
	unsigned int histh[TEMP];

	
    hipError_t cudastat;
    clock_t time1,time2,GPUtime,CPUtime;
    int size;
    size=NUM_THREADS*sizeof(unsigned int);

    time1=clock();


    //load initial values
    unsigned int* xd;
    hipMalloc((void**)&xd,size);
    hipMemcpy(xd,xtest,size,hipMemcpyHostToDevice);
	
    unsigned int* cd;
    hipMalloc((void**)&cd,size);
    hipMemcpy(cd,ctest,size,hipMemcpyHostToDevice);
	
    unsigned int* ad;
    hipMalloc((void**)&ad,size);
    hipMemcpy(ad,atest,size,hipMemcpyHostToDevice);
	
    //allocate numd on the device
	unsigned int* numd;
    hipMalloc((void**)&numd,size);

	for(i=0;i<TEMP;i++)hist[i]=0;
	unsigned int* histd;
    hipMalloc((void**)&histd,size);
	hipMemcpy(histd,hist,TEMP*sizeof(unsigned int),hipMemcpyHostToDevice);

    dim3 dimBlock(NUM_THREADS_PER_BLOCK);
    dim3 dimGrid(NUM_BLOCKS);
	hipDeviceSynchronize(); //probably not necessary


    MCd<<<dimGrid,dimBlock>>>(xd,cd,ad,numd,histd);



	hipMemcpy(num,numd,size,hipMemcpyDeviceToHost);
	hipMemcpy(hist,histd,TEMP*sizeof(unsigned int),hipMemcpyDeviceToHost);

    hipDeviceSynchronize(); //probably not necessary

    cudastat=hipGetLastError();
	printf("\nError code=%i\n",cudastat);
    printf("Error code=%i, %s.\n",cudastat,hipGetErrorString(cudastat));

	//Free device memory
    hipFree(xd);
    hipFree(cd);
    hipFree(ad);
	hipFree(numd);
	hipFree(histd);

    time2=clock();

	num_tot=0;
	for(i=0;i<NUM_THREADS;i++)num_tot+=num[i];

	hist_tot=0;
	for(i=0;i<TEMP;i++)hist_tot+=hist[i];
	for(i=0;i<TEMP;i++)printf("%d ",hist[i]);

	FILE *file;
	file = fopen("outp.txt", "w");
	for(i=0;i<TEMP;i++)fprintf(file,"%d %d\n", i, hist[i]);
	fclose(file);
	printf("\nTotal number of photons terminated (i.e. full path simulated): %llu\nNumber of photons contribution to the histogram: %llu\n",num_tot,hist_tot);
	printf("Total number of photons steps simulated: %e\n",(double)NUM_THREADS*(double)NUMSTEPS_GPU);
    printf("time1=%u, time2=%u.\n",time1,time2);

	printf("Photon steps per sec: %e\n",((double)NUM_THREADS*(double)NUMSTEPS_GPU)*CLOCKS_PER_SEC/(double(time2-time1)));
	GPUtime=time2-time1;


	printf("\n\nRunning CPU code\n");

	for(i=0;i<TEMP;i++)histh[i]=0;

	//run CPU code
	time1=clock();
	MCh(xtest,ctest,atest,numh,histh);
    time2=clock();

	num_tot=0;
	for(i=0;i<NUM_THREADS;i++)num_tot+=numh[i];

	hist_tot=0;
	for(i=0;i<TEMP;i++)hist_tot+=histh[i];
	for(i=0;i<TEMP;i++)printf("%d ",histh[i]);

	file = fopen("outph.txt", "w");
	for(i=0;i<TEMP;i++)fprintf(file,"%d ",histh[i]);
	fclose(file);
	printf("\n\nTotal number of photons (i.e. full path simulated): %llu\nNumber of photons contribution to the histogram: %llu\n",num_tot,hist_tot);
	printf("Total number of photons steps simulated: %e\n",(double)NUM_THREADS*(double)NUMSTEPS_CPU);
    printf("time1=%u, time2=%u.\n",time1,time2);

	printf("Photon steps per sec: %e\n",((double)NUM_THREADS*(double)NUMSTEPS_CPU)*CLOCKS_PER_SEC/(double(time2-time1)));
	CPUtime=time2-time1;

	printf("\n\nSpeedup: %f",(NUMSTEPS_GPU*double(CPUtime))/(NUMSTEPS_CPU*double(GPUtime)));
}




void initialize(void)//Straight from Steven Gratton's code
{
    FILE *fp;
    unsigned int begin=0u;
    unsigned long long int xinit=1ull;
    unsigned int cinit=0u;
    unsigned int fora,tmp1,tmp2;
    //fp=fopen("C:\\Users\\Jordan\\Desktop\\CUDAMC\\safeprimes_base32.txt","r");//use an expanded list containing 50000 safeprimes instead of Steven's shorter list
    fp=fopen("SRC/HelperFiles/safeprimes_base32.txt","r");//use an expanded list containing 50000 safeprimes instead of Steven's shorter list


// use begin as a multiplier to generate the initial x's for 
// the other generators...
	if(fp!=NULL)
		fscanf(fp,"%u %u %u",&begin,&tmp1,&tmp2);
	else
		printf("I'm retarded\n");

    for (unsigned int i=0;i<NUM_THREADS;i++)
    {

	xinit=xinit*begin+cinit;
	cinit=xinit>>32;
	xinit=xinit&0xffffffffull;
	xtest[i]=(unsigned int) xinit;
	fscanf(fp,"%u %u %u",&fora,&tmp1,&tmp2);
	atest[i]=fora;

	xinit=xinit*begin+cinit;
	cinit=xinit>>32;
	xinit=xinit&0xffffffffull;
	ctest[i]=(unsigned int) ((((double)xinit)/UINT_MAX)*fora);

    }
    fclose(fp);
}



int main(int argc,char* argv[])
{
	//do all the initialization for the RNG's (one MWC per thread)
    initialize();
    MC(xtest,ctest,atest);
	return 0;
    
}
